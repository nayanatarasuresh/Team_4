#include "hip/hip_runtime.h"
//libraries
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// or #include "hip/hip_runtime.h"
// #include <hip/hip_runtime_api.h>
//#include <hiprand/hiprand_kernel.h>
#include <hipsolver.h>
//#include <cusolver.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include <hipblas.h>
#include <sys/time.h>
#include "team4_readme.h"

/* Macros */
#define DATASIZE     15 //1-15

double timer_val() {
		struct timeval st;
		gettimeofday( &st, NULL );
		return (st.tv_sec+st.tv_usec*1e-6);
	} 
	
int main()
{
	int i, j;
	int data;
	int n;
	int lda = 0;
	clock_t start = 0, end = 0;
	double exec_time;
	hipsolverStatus_t status_ldl; 
	hipsolverStatus_t buff_size;
	FILE *fp = fopen("output_ldl_cuda.csv", "w"); 
	fclose(fp);
	
	// Running the LDLT for matrices of different sizes
	for (data = 1; data <= DATASIZE; data++) {
		n = exp2((double)data);
		
		double *A = (double*)malloc(n*n*sizeof(double));
	
		for (i = 0; i < n; i++) {
			for (j = 0; j < n; j++) {
				
				if ( i == j) {
					*((A + i) + n*j) = i + ( (j + 1) * (3 + i));
				}
				else {
					*((A + i) + n*j) = i + j ;
				}
				
			}
			
		}
		
		
		
		//allocating memory on the GPU by copying the matrix here
		double *Z;
		hipMalloc(&Z, n*n*sizeof(double));		
		hipMemcpy(Z, A, n*n*sizeof(double), hipMemcpyHostToDevice);
		
		//enabling the handle
		hipsolverHandle_t handle;	
    	hipsolverDnCreate(&handle);
		
		// 
		int *devInfo; 
		hipMalloc(&devInfo, sizeof(int)); // ask if this needs allocation of memory
		
		lda = n;
		int sizeof_work = 0;
		buff_size = hipsolverDnDsytrf_bufferSize(handle, n, Z, lda, &sizeof_work);	
		
		double *work;
		hipMalloc(&work, sizeof_work * sizeof(double));
		int *ipiv;
		hipMalloc(&ipiv, n*sizeof(int));
		
		start = timer_val();
		//start = cusolver_test_seconds();
		for(j=0; j<1000; j++)	
		{
			//hipsolverDnDsytrf(hipsolverHandle_t handle,hipblasFillMode_t uplo,int n,double *A,int lda,int *ipiv,double *work,int lwork,int *devInfo );
		
			// Computing the LDLT decomposition
			status_ldl = hipsolverDnDsytrf(handle, HIPBLAS_FILL_MODE_LOWER, n, Z, lda, ipiv, work, sizeof_work, devInfo);
			
			printf ("\n The device info is %d\n", devInfo);
			
			if ( status_ldl == HIPSOLVER_STATUS_SUCCESS ) {
				printf ("\nThe status of the LDLT decomposition is HIPSOLVER_STATUS_SUCCESS");
			}
			else if ( status_ldl == HIPSOLVER_STATUS_NOT_INITIALIZED ) {
				printf ("\nThe status of the LDLT decomposition is HIPSOLVER_STATUS_NOT_INITIALIZED");
			}
			else if ( status_ldl == HIPSOLVER_STATUS_INVALID_VALUE ) {
				printf ("\nThe status of the LDLT decomposition is HIPSOLVER_STATUS_INVALID_VALUE");
			}
			else if ( status_ldl == HIPSOLVER_STATUS_ARCH_MISMATCH ) {
				printf ("\nThe status of the LDLT decomposition is HIPSOLVER_STATUS_ARCH_MISMATCH");
			}
			else if ( status_ldl == HIPSOLVER_STATUS_INTERNAL_ERROR ) {
				printf ("\nThe status of the LDLT decomposition is HIPSOLVER_STATUS_INTERNAL_ERROR");
			}
		} 
		end = timer_val();
		//end = cusolver_test_seconds();
		
		exec_time = (double)(end - start);
		//exec_time = exec_time;                         //dividing by 1000 because we have executed 1000 iterations
		
		printf ("The execution for LDLT decomposition of a %d by %d matrix is %.15f s\n", n, n, exec_time/1000.00);
		//printf ("The execution for LDLT decomposition of a %d by %d matrix is %.8f s\n", n, n, exec_time*1000);
			
		// open file where the output should be saved
		FILE *fp = fopen("output_ldl_cuda.csv", "a"); 
		fprintf(fp, "The datasize is %d\n", n);
		fprintf(fp, "The execution for LDLT decomposition of a %d by %d matrix is %.15f s\n", n, n, exec_time/1000.00);
		fclose(fp);
		
		free(A);
		hipsolverDnDestroy(handle);
		hipFree(Z);
	}
}	


/*things to ask
ask if devInfo needs allocation of memory and the cmds to exxecute cuda 
cusolver_test_seconds not working
*/
